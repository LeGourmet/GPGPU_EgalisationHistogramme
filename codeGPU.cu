#include "hip/hip_runtime.h"
#include "codeGPU.hpp"
#include "utils/chrono/common.hpp"
#include "utils/chrono/chronoGPU.hpp"


// ********************************************* RGB2HSV *********************************************
__global__ void computeRGB2HSV(const unsigned char* data, const int sizeData, float* const hsv) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	float cmax, cmin, delta, r, g, b, h;
	while(i<sizeData){
		
		r = data[i*3] / 255.0f;
		g = data[i*3+1] / 255.0f;
		b = data[i*3+2] / 255.0f;
		
		cmax = fmax(r, fmax(g, b));
		cmin = fmin(r, fmin(g, b));
		delta = cmax - cmin;
		
		hsv[i*3+2] = cmax;
		
		if(cmax == 0.0f) {
			h = hsv[i*3+1] = 0.0f;
		} else {
			hsv[i*3+1] = delta / cmax;
			if(delta < 0.001f) {
				h = 0.0f;
			} else {
				if(cmax == r) {
					h = 60.0f * (g - b)/delta;
					if(h < 0.0f) { h += 360.0f; }
				} else if(cmax == g) {
					h = 60.0f * (2 + (b - r)/delta);
				} else {
					h = 60.0f * (4 + (r - g)/delta);
				}
			}		
		}
		hsv[i*3] = h;
		i += blockDim.x * gridDim.x;
	}
}

float rgb2hsv_GPU(Image* img, HSV* hsv){
	// data device
	unsigned char* dev_inPtr;
	float* dev_outPtr;

	float* resultArr = new float[hsv->size*3];

	unsigned long sData = hsv->size*3*sizeof(unsigned char);
	unsigned long sRes = hsv->size*3*sizeof(float);

	ChronoGPU chr;
	chr.start();

	// Allocate memory on Device
		HANDLE_ERROR(hipMalloc(&dev_inPtr, sData));
		HANDLE_ERROR(hipMalloc(&dev_outPtr, sRes));

	// Copy from Host to Device
		HANDLE_ERROR(hipMemcpy(dev_inPtr, img->_pixels, sData, hipMemcpyHostToDevice));

	// Launch kernel
		computeRGB2HSV <<< 1024, 256 >>>(dev_inPtr,hsv->size,dev_outPtr);

	// Copy from Device to Host
		HANDLE_ERROR(hipMemcpy(resultArr, dev_outPtr, sRes, hipMemcpyDeviceToHost));

	// Free memory on Device
		HANDLE_ERROR(hipFree(dev_outPtr));
		HANDLE_ERROR(hipFree(dev_inPtr));

	// Delinearize data
	for(int i=0;i<hsv->size;i++){
		hsv->H[i] = resultArr[i*3];
		hsv->S[i] = resultArr[i*3+1];
		hsv->V[i] = resultArr[i*3+2];
	}

	chr.stop();

	return chr.elapsedTime();
}

// ********************************************* HSV2RGB *********************************************
__global__ void computeHSV2RGB(const float* data, const int sizeData, unsigned char* const pixels) {
	//basé sur https://docs.nvidia.com/cuda/npp/group__hsvtorgb.html
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	float h, s, v, h2, frac, m, n, k, r, b, g;
	while(i<sizeData){
		h = data[i*3];
		s = data[i*3+1];
		v = data[i*3+2];

		h2 = h/60.f; 
		frac = h2 - floorf(h2);
		m = v * (1.0f - s);  			 
		n = v * (1.0f - s * frac);	 
		k = v * (1.0f - s * (1.0f - frac));
		
		if (h2>=0 && h2<1.f) {
			r = v;g = k;b = m;
		} else if (h2<2.f) {
			r = n;g = v;b = m;
		} else if (h2<3.f) {
			r = m;g = v;b = k;
		} else if (h2<4.f) {
			r = m;g = n;b = v;
		} else if (h2<5.f) {
			r = k;g = m;b = v;
		} else if (h2<6.f){
			r = v;g = m;b = n;
		}
		
		pixels[i*3] = r*255.f;
		pixels[i*3+1] = g*255.f;
		pixels[i*3+2] = b*255.f;
		i += blockDim.x * gridDim.x;
	}
}

float hsv2rgb_GPU(HSV* hsv, Image* img){
	// data device
	float* dev_inPtr;
	unsigned char* dev_outPtr;

	unsigned long sData = hsv->size*3*sizeof(float);
	unsigned long sRes = hsv->size*3*sizeof(unsigned char);

	// linearize data
	float* data = new float[hsv->size*3];
	for(int i = 0;i<hsv->size;i++){
		data[i*3] = hsv->H[i];
		data[i*3+1] = hsv->S[i];
		data[i*3+2] = hsv->V[i];
	}

	ChronoGPU chr;
	chr.start();

	// Allocate memory on Device
		HANDLE_ERROR(hipMalloc(&dev_inPtr, sData));
		HANDLE_ERROR(hipMalloc(&dev_outPtr, sRes));

	// Copy from Host to Device
		HANDLE_ERROR(hipMemcpy(dev_inPtr, data, sData, hipMemcpyHostToDevice));

	// Launch kernel
		computeHSV2RGB <<< 512, 1024 >>>(dev_inPtr,hsv->size,dev_outPtr);
	
	// Copy from Device to Host
		HANDLE_ERROR(hipMemcpy(img->_pixels, dev_outPtr, sRes, hipMemcpyDeviceToHost));

	// Free memory on Device
		HANDLE_ERROR(hipFree(dev_outPtr));
		HANDLE_ERROR(hipFree(dev_inPtr));

	chr.stop();


	return chr.elapsedTime();
}

// ********************************************* HISTO *********************************************
__global__ void computeHisto(const float* data, const int sizeData, int* const histo) {
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	while(i<sizeData){
		int j = data[i]*255.f;
		atomicAdd(&histo[j],1);
		i += blockDim.x * gridDim.x;
	}
}

float histogram_GPU(HSV* hsv, int* histo) {
	// data device
	float* dev_inPtr;
	int* dev_outPtr;
	
	unsigned long sData = hsv->size*sizeof(float);
	unsigned long sHisto =  256*sizeof(int);

	ChronoGPU chr;
	chr.start();

	// Allocate memory on Device
		HANDLE_ERROR(hipMalloc(&dev_inPtr, sData));
		HANDLE_ERROR(hipMalloc(&dev_outPtr, sHisto));

	// Copy from Host to Device
		HANDLE_ERROR(hipMemcpy(dev_inPtr, hsv->V, sData, hipMemcpyHostToDevice));

	// Launch kernel
		computeHisto <<< 512, 1024 >>>(dev_inPtr,hsv->size,dev_outPtr);

	// Copy from Device to Host
		HANDLE_ERROR(hipMemcpy(histo, dev_outPtr, sHisto, hipMemcpyDeviceToHost));

	// Free memory on Device
		HANDLE_ERROR(hipFree(dev_outPtr));
		HANDLE_ERROR(hipFree(dev_inPtr));

	chr.stop();

	return chr.elapsedTime();
}

// ********************************************* REPARTITION *********************************************
__global__ void computeRepart( const int* histo, int* const repart ){
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	int sum = 0;
	int nbElem = 2;

	while( i < 256 ){
		repart[i]=histo[i];
		__syncthreads();
		while(nbElem<=256){
			if(i*nbElem>256){break;}
			sum = repart[i*nbElem+(nbElem/2)-1];
			for(int j=nbElem/2; j<nbElem ;j++){
				repart[i*nbElem+j] += sum;
			}
			nbElem *= 2;
		}

		i += blockDim.x * gridDim.x;;
	}
}

float repart_GPU(int* histo, int* repart){
	// data device
	int* dev_inPtr;
	int* dev_outPtr;
	
	unsigned long size = 256*sizeof(int);

	ChronoGPU chr;
	chr.start();

	// Allocate memory on Device
		HANDLE_ERROR(hipMalloc(&dev_inPtr, size));
		HANDLE_ERROR(hipMalloc(&dev_outPtr, size));

	// Copy from Host to Device
		HANDLE_ERROR(hipMemcpy(dev_inPtr, histo, size, hipMemcpyHostToDevice));

	// Launch kernel
		computeRepart <<< 512, 1024 >>>(dev_inPtr,dev_outPtr);

	// Copy from Device to Host
		HANDLE_ERROR(hipMemcpy(repart, dev_outPtr, size, hipMemcpyDeviceToHost));

	// Free memory on Device
		HANDLE_ERROR(hipFree(dev_outPtr));
		HANDLE_ERROR(hipFree(dev_inPtr));

		chr.stop();

	return chr.elapsedTime();
}

// ********************************************* EQUALISATION *********************************************
__global__ void computeEqualization( const int *repart, const int sizeV, const float* V_in  ,float* const V_out){
 	int i = threadIdx.x + blockIdx.x*blockDim.x;
	while(i<sizeV){
		V_out[i] = ((255.f/(sizeV))*repart[(int)(V_in[i]*255.f)])/255.f;
		i += blockDim.x * gridDim.x;
	}
}

float equalization_GPU(HSV *hsv, int* repart){
	// data device
	int* repart_inPtr;
	float* V_inPtr;
	float* V_outPtr;
	
	unsigned long sRepart = 256*sizeof(int);
	unsigned long sV =      hsv->size*sizeof(float);

	ChronoGPU chr;
	chr.start();
	// Allocate memory on Device
		HANDLE_ERROR(hipMalloc(&repart_inPtr, sRepart));
		HANDLE_ERROR(hipMalloc(&V_inPtr, sV));
		HANDLE_ERROR(hipMalloc(&V_outPtr, sV));

	// Copy from Host to Device
		HANDLE_ERROR(hipMemcpy(repart_inPtr, repart, sRepart, hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(V_inPtr, hsv->V, sV, hipMemcpyHostToDevice));

	// Launch kernel
		computeEqualization <<< 128, 1024 >>>(repart_inPtr,hsv->size,V_inPtr,V_outPtr);

	// Copy from Device to Host
		HANDLE_ERROR(hipMemcpy(hsv->V, V_outPtr, sV, hipMemcpyDeviceToHost));

	// Free memory on Device
		HANDLE_ERROR(hipFree(V_outPtr));
		HANDLE_ERROR(hipFree(V_inPtr));
		HANDLE_ERROR(hipFree(repart_inPtr));

		chr.stop();

	return chr.elapsedTime();
}